
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define DIM 512

struct Lock {
  int *mutex;
  Lock( void ) {
    int state = 0;
    hipMalloc( (void**)& mutex, sizeof(int) );
    hipMemcpy( mutex, &state, sizeof(int), hipMemcpyHostToDevice );
  }

  ~Lock( void ) {
    hipFree( mutex );
  }

  __device__ void lock( void ) {
    while( atomicCAS( mutex, 0, 1 ) != 0 );
  }

  __device__ void unlock( void ) {
   atomicExch( mutex, 0 );
  }
};

// Forward function declarations
float CPU_big_dot(float *A, float *B, int N);
void GPU_big_dot(float *A, float *B, int N);
float *get_random_vector(int N);
void die(const char *message);

template <unsigned int iBlockSize>
__global__ void kernel1(float *g_idata, float *g_odata, unsigned int N)
{
    __shared__ float smem[DIM];

    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    float *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8
    if (idx + 7 * blockDim.x < N)
    {
        float a1 = g_idata[idx];
        float a2 = g_idata[idx + blockDim.x];
        float a3 = g_idata[idx + 2 * blockDim.x];
        float a4 = g_idata[idx + 3 * blockDim.x];
        float b1 = g_idata[idx + 4 * blockDim.x];
        float b2 = g_idata[idx + 5 * blockDim.x];
        float b3 = g_idata[idx + 6 * blockDim.x];
        float b4 = g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;
    }
    smem[tid] = idata[tid];
    __syncthreads();

    // in-place reduction and complete unroll
    if (iBlockSize >= 1024 && tid < 512) smem[tid] += smem[tid + 512];
    __syncthreads();

    if (iBlockSize >= 512 && tid < 256) smem[tid] += smem[tid + 256];
    __syncthreads();

    if (iBlockSize >= 256 && tid < 128) smem[tid] += smem[tid + 128];
    __syncthreads();

    if (iBlockSize >= 128 && tid < 64) smem[tid] += smem[tid + 64];
    __syncthreads();

    // unrolling warp
    if (tid < 32)
    {
        volatile float *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}

template <unsigned int iBlockSize>
__global__ void kernel2(float *g_idata, float *C, unsigned int N)
{
    __shared__ float smem[DIM];

    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    float *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8
    if (idx + 7 * blockDim.x < N)
    {
        float a1 = g_idata[idx];
        float a2 = g_idata[idx + blockDim.x];
        float a3 = g_idata[idx + 2 * blockDim.x];
        float a4 = g_idata[idx + 3 * blockDim.x];
        float b1 = g_idata[idx + 4 * blockDim.x];
        float b2 = g_idata[idx + 5 * blockDim.x];
        float b3 = g_idata[idx + 6 * blockDim.x];
        float b4 = g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;
    }
    smem[tid] = idata[tid];
    __syncthreads();

    // in-place reduction and complete unroll
    if (iBlockSize >= 1024 && tid < 512) smem[tid] += smem[tid + 512];
    __syncthreads();

    if (iBlockSize >= 512 && tid < 256) smem[tid] += smem[tid + 256];
    __syncthreads();

    if (iBlockSize >= 256 && tid < 128) smem[tid] += smem[tid + 128];
    __syncthreads();

    if (iBlockSize >= 128 && tid < 64) smem[tid] += smem[tid + 64];
    __syncthreads();

    // unrolling warp
    if (tid < 32)
    {
        volatile float *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) atomicAdd(C, smem[0]);
}

int main(int argc, char **argv) {
        // Seed the random generator (use a constant here for repeatable results)
        srand(10);

        // Determine the vector length
        int N = 1 << 24;

        // Generate two random vectors
        float *A= get_random_vector(N);
        float *B= get_random_vector(N);

        // Compute their dot product on the CPU
        float sumCPU = CPU_big_dot(A, B, N);
        printf("dot product with CPU= %f\n", sumCPU);
        
        // Compute their dot product on the GPU
        GPU_big_dot(A, B, N);

        free(A);
        free(B);
}

// Returns the vector dot product of A and B
float CPU_big_dot(float *A, float *B, int N) {
        // Compute the dot product
        float sum = 0;
        for (int i = 0; i < N; i++) sum += A[i] * B[i];

        // Return the result
        return sum;
}

// A GPU kernel that computes the vector dot product of A and B
// (each thread computes a single value of the result)
__global__ void dot_product_multiplication_kernel(float *A, float *B, float *C, int N) {
        // Determine which element this thread is computing
        int thread_id = threadIdx.x + blockIdx.x * blockDim.x;

        // Compute a single element of the result vector (if the element is valid)
        if (thread_id < N) C[thread_id] = A[thread_id] * B[thread_id];
}

// Returns the vector dot product of A and B (computed on the GPU)
void GPU_big_dot(float *A_CPU, float *B_CPU, int N) {

        // execution configuration
        int blocksize = DIM;   // initial block size

        dim3 block (blocksize, 1);
        dim3 grid  ((N + block.x - 1) / block.x, 1);
        printf("grid %d block %d\n", grid.x, block.x);

        // Allocate GPU memory for the inputs and the result
        int vector_size = N * sizeof(float);
        float *A_GPU = NULL;
        float *B_GPU = NULL;
        float *C_GPU = NULL;
        float *O_GPU = NULL;

        if (hipMalloc((void **) &A_GPU, vector_size) != hipSuccess) die("Error allocating GPU memory");
        if (hipMalloc((void **) &B_GPU, vector_size) != hipSuccess) die("Error allocating GPU memory");
        if (hipMalloc((void **) &C_GPU, vector_size) != hipSuccess) die("Error allocating GPU memory");
        if (hipMalloc((void **) &O_GPU, grid.x * sizeof(float)) != hipSuccess) die("Error allocating GPU memory");

        // Transfer the input vectors to GPU memory
        hipMemcpy(A_GPU, A_CPU, vector_size, hipMemcpyHostToDevice);
        hipMemcpy(B_GPU, B_CPU, vector_size, hipMemcpyHostToDevice);

        // Execute the kernel to compute the vector dot product multiplication part on the GPU
        dot_product_multiplication_kernel<<< grid, block >>> (A_GPU, B_GPU, C_GPU, N);
        hipDeviceSynchronize();  // this is only needed for timing purposes

        // Check for kernel errors
        hipError_t error = hipGetLastError();
        if (error) {
          char message[256];
          sprintf(message, "CUDA error: %s", hipGetErrorString(error));
          die(message);
        }

        float *C_GPU_saved = NULL;
        if (hipMalloc((void **) &C_GPU_saved, vector_size) != hipSuccess) die("Error allocating GPU memory");
        hipMemcpy(C_GPU_saved, C_GPU, vector_size, hipMemcpyDeviceToDevice);

        // Allocate CPU memory for the result
        float *O_CPU = (float *) malloc(grid.x * sizeof(float));
        if (O_CPU == NULL) die("Error allocating CPU memory");

        hipEvent_t start, stop;
        float kernel1_time;
        hipEventCreate( &start);
        hipEventCreate( &stop);
        hipEventRecord( start, 0);
        // Execute the kernel1 to compute the vector dot product on the GPU
        switch (blocksize) {
          case 1024:
            kernel1<1024><<<grid.x/8, block>>>(C_GPU, O_GPU, N);
            break;
          case 512:
            kernel1<512><<<grid.x/8, block>>>(C_GPU, O_GPU, N);
            break;
          case 256:
            kernel1<256><<<grid.x/8, block>>>(C_GPU, O_GPU, N);
            break;
          case 128:
            kernel1<128><<<grid.x/8, block>>>(C_GPU, O_GPU, N);
            break;
          case 64:
            kernel1<64><<<grid.x/8, block>>>(C_GPU, O_GPU, N);
            break;
        }
        hipDeviceSynchronize();
        hipEventRecord( stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime( &kernel1_time, start, stop);
        printf("Time takes for kernel1: %f ms\n", kernel1_time);

        // Check for kernel errors
        hipGetLastError();
        if (error) {
          char message[256];
          sprintf(message, "CUDA error: %s", hipGetErrorString(error));
          die(message);
        }

        hipMemcpy(O_CPU, O_GPU, grid.x * sizeof(int), hipMemcpyDeviceToHost);
        float sumKernel1 = 0;
        for (int i = 0; i < grid.x / 8; i++) sumKernel1 += O_CPU[i];
        printf("Dot product with kernel1 = %f\n", sumKernel1);

        float sumKernel2 = 0;
        float *sumKernel2_GPU = NULL;
        if (hipMalloc((void **) &sumKernel2_GPU, sizeof(float)) != hipSuccess) die("Error allocating GPU memory");
        hipMemcpy(sumKernel2_GPU, &sumKernel2, sizeof(float), hipMemcpyHostToDevice);
        float kernel2_time;
        hipEventRecord( start, 0);
        // Execute the kernel2 to compute the vector dot product on the GPU
        switch (blocksize) {
          case 1024:
            kernel2<1024><<<grid.x/8, block>>>(C_GPU_saved, sumKernel2_GPU, N);
            break;
          case 512:
            kernel2<512><<<grid.x/8, block>>>(C_GPU_saved, sumKernel2_GPU, N);
            break;
          case 256:
            kernel2<256><<<grid.x/8, block>>>(C_GPU_saved, sumKernel2_GPU, N);
            break;
          case 128:
            kernel2<128><<<grid.x/8, block>>>(C_GPU_saved, sumKernel2_GPU, N);
            break;
          case 64:
            kernel2<64><<<grid.x/8, block>>>(C_GPU_saved, sumKernel2_GPU, N);
            break;
        }
        hipDeviceSynchronize();
        hipEventRecord( stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime( &kernel2_time, start, stop);
        printf("Time takes for kernel2: %f ms\n", kernel2_time);

        // Check for kernel errors
        hipGetLastError();
        if (error) {
            char message[256];
            sprintf(message, "CUDA error: %s", hipGetErrorString(error));
            die(message);
        }

        // Transfer the result from teh GPU to the CPU
        hipMemcpy(&sumKernel2, sumKernel2_GPU, sizeof(float), hipMemcpyDeviceToHost);
        printf("Dot product with kernel2 = %f\n", sumKernel2);

        // Compute the speedup or slowdown
        if (kernel1_time > kernel2_time) printf("kernel1 outperformed kernel2 by %.2fx\n", (float) kernel2_time / (float) kernel1_time);
        else printf("kernel2 outperformed kernel1 by %.2fx\n", (float) kernel1_time / (float) kernel2_time);

        hipFree(A_GPU);
        hipFree(B_GPU);
        hipFree(C_GPU);
        hipFree(O_GPU);
        hipFree(sumKernel2_GPU);

        free(O_CPU);
}

// Returns a randomized vector containing N elements
float *get_random_vector(int N) {
        if (N < 1) die("Number of elements must be greater than zero");

        // Allocate memory for the vector
        float *V = (float *) malloc(N * sizeof(float));
        if (V == NULL) die("Error allocating CPU memory");

        // Populate the vector with random numbers
        for (int i = 0; i < N; i++) V[i] = (float) rand() / (float) rand();

        // Return the randomized vector
        return V;
}

// Prints the specified message and quits
void die(const char *message) {
        printf("%s\n", message);
        exit(1);
}
